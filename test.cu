#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
using namespace std;
#include "book.h"
__global__ void add(int a, int b, int* c) {
    *c = a + b;
}



int main(){
    /*
    int count;
    HANDLE_ERROR(hipGetDeviceCount( &count ));
    cout << "Count: " << count << endl;
    hipDeviceProp_t prop;
    for(int i = 0; i < count; i++) {
        HANDLE_ERROR(hipGetDeviceProperties( &prop, i));
        cout << "name: " << prop.name << endl;
    }
    */

    hipDeviceProp_t prop;
    int dev;

    HANDLE_ERROR(hipGetDevice(&dev));
    printf("ID of current CUDA device: %d\n", dev);

    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = 1;
    prop.minor = 3;
    HANDLE_ERROR(hipChooseDevice(&dev, &prop));
    printf("ID of CUDA device closest to revision 1.3: %d\n", dev);

    HANDLE_ERROR(hipGetDevice(&dev));
    int c;
    int* dev_c;
    HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int)));
    add<<<1, 1>>>(2, 7, dev_c);
    HANDLE_ERROR(hipMemcpy(&c,
                            dev_c,
                            sizeof(int),
                            hipMemcpyDeviceToHost));

    printf("2 + 7 = %d\n", c);
    hipFree(dev_c);
    return 0;
}
